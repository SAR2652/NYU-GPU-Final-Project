/* 
 * This file contains the code for doing the heat distribution problem. 
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s), memory allocation, data movement, etc. 
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void seq_heat_dist(float *, unsigned int, unsigned int);
void gpu_heat_dist(float *, unsigned int, unsigned int);
__global__ void heat_distribution_kernel(float*, float*, unsigned int);

/*****************************************************************/
/**** Do NOT CHANGE ANYTHING in main() function ******/

int main(int argc, char * argv[])
{
  unsigned int N; /* Dimention of NxN matrix */
  int type_of_device = 0; // CPU or GPU
  int iterations = 0;
  int i;
  
  /* The 2D array of points will be treated as 1D array of NxN elements */
  float * playground; 
  
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  if(argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
    exit(1);
  }
  
  type_of_device = atoi(argv[3]);
  N = (unsigned int) atoi(argv[1]);
  iterations = (unsigned int) atoi(argv[2]);
 
  
  /* Dynamically allocate NxN array of floats */
  playground = (float *)calloc(N*N, sizeof(float));
  if( !playground )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
   exit(1);
  }
  
  /* Initialize it: calloc already initalized everything to 0 */
  // Edge elements  initialization
  for(i = 0; i < N; i++)
    playground[index(0,i,N)] = 100;
  for(i = 0; i < N-1; i++)
    playground[index(N-1,i,N)] = 150;

  if( !type_of_device ) // The CPU sequential version
  {  
    start = clock();
    seq_heat_dist(playground, N, iterations);
    end = clock();
  }
  else  // The GPU version
  {
     start = clock();
     gpu_heat_dist(playground, N, iterations); 
     end = clock();    
  }
  
  
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  
  printf("Time taken for %s is %lf\n", type_of_device == 0? "CPU" : "GPU", time_taken);
  
  free(playground);
  
  return 0;

}


/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void  seq_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
  // Loop indices
  int i, j, k;
  int upper = N-1;
  
  // number of bytes to be copied between array temp and array playground
  unsigned int num_bytes = 0;
  
  float * temp; 
  /* Dynamically allocate another array for temp values */
  /* Dynamically allocate NxN array of floats */
  temp = (float *)calloc(N*N, sizeof(float));
  if( !temp )
  {
   fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
   exit(1);
  }
  
  num_bytes = N*N*sizeof(float);
  
  /* Copy initial array in temp */
  memcpy((void *)temp, (void *) playground, num_bytes);
  
  for( k = 0; k < iterations; k++)
  {
    /* Calculate new values and store them in temp */
    for(i = 1; i < upper; i++)
      for(j = 1; j < upper; j++)
	temp[index(i,j,N)] = (playground[index(i-1,j,N)] + 
	                      playground[index(i+1,j,N)] + 
			      playground[index(i,j-1,N)] + 
			      playground[index(i,j+1,N)])/4.0;
  
			      
   			      
    /* Move new values into old values */ 
    memcpy((void *)playground, (void *) temp, num_bytes);
  }

 // printf("Host Playground\n");
   // for(unsigned int i = 0; i < N; i++)
   // {
       // for(unsigned int j = 0; j < N; j++)
       // {
       //     printf("%f ", playground[index(i, j, N)]);
       // }
       // printf("\n");
    // }
  
}

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kernels if you want ********************/
void  gpu_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
    unsigned long long grid_count = pow(N, 2);
    unsigned long long alloc_size = grid_count * sizeof(float);
    float *device_playground, *intermediate_storage;

    hipMalloc((void**) &device_playground, alloc_size);
    hipMalloc((void**) &intermediate_storage, alloc_size);

    hipMemcpy(device_playground, playground, alloc_size, hipMemcpyHostToDevice);
    hipMemcpy(intermediate_storage, playground, alloc_size, hipMemcpyHostToDevice);

    unsigned int block_dim = 16;
    unsigned int grid_dim = ceil((double) N / block_dim);
    
    dim3 block_var(block_dim, block_dim, 1);
    dim3 grid_var(grid_dim, grid_dim, 1);

    for(unsigned int i = 0; i < iterations; i++)
    {
        heat_distribution_kernel<<<block_var, grid_var>>>(device_playground, intermediate_storage, N);
	hipMemcpy(device_playground, intermediate_storage, alloc_size, hipMemcpyDeviceToDevice);
    }

    hipMemcpy(playground, device_playground, alloc_size, hipMemcpyDeviceToHost);

    // printf("Device Playground\n");
    // for(unsigned int i = 0; i < N; i++)
    // {
       //  for(unsigned int j = 0; j < N; j++)
        // {
           //  printf("%f ", playground[index(i, j, N)]);
        // }
        // printf("\n");
    // }

    hipFree(device_playground);
    hipFree(intermediate_storage);
}

__global__ void heat_distribution_kernel(float* device_playground, float* intermediate_storage, unsigned int N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

    bool vertical_lower_limit = i >= 1;
    bool horizontal_lower_limit = j >= 1;
    bool vertical_upper_limit = i <= N - 2;
    bool horizontal_upper_limit = j <= N - 2;

    if(vertical_lower_limit && horizontal_lower_limit && horizontal_upper_limit && vertical_upper_limit)
    {
        intermediate_storage[index(i, j, N)] = device_playground[index(i - 1, j, N)] +
                                                device_playground[index(i, j - 1, N)] +
                                                device_playground[index(i + 1, j, N)] +
                                                device_playground[index(i, j + 1, N)];

        intermediate_storage[index(i, j, N)] /= 4.0;
    }
}


