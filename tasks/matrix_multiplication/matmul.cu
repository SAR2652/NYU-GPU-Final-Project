#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds


__global__ void matMul(float* Pd, float* Md, float* Nd, int Width, int Tile_Width) {
  float Pvalue = 0.0;

  int j = blockIdx.x * Tile_Width + threadIdx.x;
  int i = blockIdx.y * Tile_Width + threadIdx.y;
if (j < Width && i < Width) {
  for (int k = 0; k < Width; ++k) {
    Pvalue += Md[i * Width + k] * Nd[k * Width + j];
  }
}
  Pd[i * Width + j] = Pvalue;
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

// void seq_mat_mul(float* M, float* N, float* P, int Width)
// {
//     for(int i = 0; i < Width; i++)
//     {
//         for(int j = 0; j < Width; j++)
//         {
//             for(int k = 0; k < Width; k++)
//             {
                
//             }
//         }
//     }
// }

int main(int argc, char* argv[])
{
  clock_t start, end;

  int Width = atoi(argv[1]);
  int Tile_Width = atoi(argv[2]);
  //int devId = atoi(argv[3]);

  //checkCuda( cudaSetDevice(devId) );
  //cudaDeviceReset();

  // allocate host memory for matrices M and N
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));
  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  start = clock();

  hipMalloc((void**) &Md, Width * Width * sizeof(float));
  hipMalloc((void**) &Nd, Width * Width * sizeof(float));
  hipMalloc((void**) &Pd, Width * Width * sizeof(float));

  // copy host memory to device
  hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Pd, P, Width*Width*sizeof(float), hipMemcpyHostToDevice);

  // execute the kernel

  int GridSize = (Width + Tile_Width-1) / Tile_Width;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);
  
  matMul<<< gridDim, blockDim >>>(Pd, Md, Nd, Width,Tile_Width);

  // copy result from device to host
  hipMemcpy(P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost);

  // clean up memory  
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);

  end = clock();
  double time_taken = (double)(end - start) / CLOCKS_PER_SEC;
  printf("%lf\n%d\n", time_taken, GridSize);

  free(M);
  free(N);
  free(P);

  return 0;
}
