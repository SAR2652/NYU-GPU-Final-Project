#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float *, float *, float *, int,int);

/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	
		
	n = atoi(argv[1]);
	int numblocks = atoi(argv[2]);
	int num_threads = atoi(argv[3]);
	//printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   //printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   //printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   //printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   //printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
	int threadsperblock;
	int THREADS = num_threads;
	threadsperblock = THREADS;
	int num_comps = ceil((double)n/(numblocks*threadsperblock));
	
    //The sequential part
	//start = clock();
	//for(i = 0; i < n; i++)
	//	temp[i] += a[i] * b[i];
	//end = clock();
	//printf("%lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/

	
	hipMalloc((void **)&ad, n*sizeof(float));

	hipMalloc((void **)&bd, n*sizeof(float));
	
	hipMalloc((void **)&cd, n*sizeof(float));
	hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
	
	hipMemcpy(bd, b, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cd, c, n*sizeof(float), hipMemcpyHostToDevice);
	vecGPU<<<numblocks, threadsperblock>>> (ad,bd,cd,n,num_comps);
	start = clock();
	hipMemcpy(c, cd, n*sizeof(float), hipMemcpyDeviceToHost);
	end = clock();
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	printf("%lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	//for(i = 0; i < n; i++)
	  //if( abs(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		//printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);
	return 0;
}



/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(float * a, float * b, float *c, int n, int num_threads_percomp)
{


		int id  = (blockIdx.x * blockDim.x) + threadIdx.x;
		
		int start_id = id*num_threads_percomp;
		int end_id = num_threads_percomp + start_id;
		if(start_id < n)
		{
			for( int i = start_id; i< end_id; i++)
			{	if(i >= n)
				   break;
				c[i] += a[i] * b[i];
			}
		}
		

		
}
