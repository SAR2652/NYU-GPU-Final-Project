#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.



__global__ void matSum(float* S, float* A, float* B, int N) {
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int tid = i*N + j;
  if (tid < N*N) {
    S[tid] = A[tid] + B[tid];
  }
}


// Fills a vector with random float entries.
void randomInit(float* data, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      int tid = i*N+j;
      data[tid] = (float)drand48();
    }
  }
}


int main(int argc, char* argv[])
{

//   if (argc != 3) {
//     // fprintf(stderr, "Syntax: %s <matrix size> <CacheConfL1>  <device> \n", argv[0]);
//     return EXIT_FAILURE;
//   }

  clock_t start, end; 
  int N = atoi(argv[1]);
  int Tile_Width = atoi(argv[3]);
  //int devId = atoi(argv[2]);

  //cudaDeviceReset();

  // set seed for drand48()
  srand48(42);

  // allocate host memory for matrices A and B
  float* A = (float*) malloc(N * N * sizeof(float));
  float* B = (float*) malloc(N * N * sizeof(float));
  float* S = (float*) malloc(N * N * sizeof(float));

  // initialize host matrices
  
  randomInit(A, N);
  randomInit(B, N);

  // allocate device matrices (linearized)
  float* dev_A = NULL; 
  float* dev_B = NULL;
  float* dev_S = NULL;
  hipMalloc((void**) &dev_A, N * N * sizeof(float));
  hipMalloc((void**) &dev_B, N * N * sizeof(float));
  hipMalloc((void**) &dev_S, N * N * sizeof(float));

  // copy host memory to device
  hipMemcpy(dev_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);

  // execute the kernel

  int GridSize = atoi(argv[2]);
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);
  start = clock();
  matSum<<< gridDim, blockDim >>>(dev_S, dev_A, dev_B, N);
  end = clock();

  // copy result from device to host
  hipMemcpy( S, dev_S, N * N * sizeof(float),hipMemcpyDeviceToHost);

  //cudaDeviceProp prop;
  //cudaGetDeviceProperties(&prop, devId);
  //printf("Device: %s\n", prop.name);

  // clean up memory
  free(A);
  free(B);
  free(S);
  hipFree(dev_A);
  hipFree(dev_B);
  hipFree(dev_S);
  printf("%.8f\n", (double)(end - start) / CLOCKS_PER_SEC);
  return 0;
}