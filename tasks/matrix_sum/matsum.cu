#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.



__global__ void matSum(float* S, float* A, float* B, int N) {
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int tid = i*N + j;
  if (tid < N*N) {
    S[tid] = A[tid] + B[tid];
  }
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

int main(int argc, char* argv[])
{
  clock_t start, end;

  int Width = atoi(argv[1]);
  int Tile_Width = atoi(argv[3]);
  //int devId = atoi(argv[3]);

  //checkCuda( cudaSetDevice(devId) );
  //cudaDeviceReset();

  // allocate host memory for matrices M and N
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));
  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  start = clock();

  hipMalloc((void**) &Md, Width * Width * sizeof(float));
  hipMalloc((void**) &Nd, Width * Width * sizeof(float));
  hipMalloc((void**) &Pd, Width * Width * sizeof(float));

  // copy host memory to device
  hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Pd, P, Width*Width*sizeof(float), hipMemcpyHostToDevice);

  // execute the kernel

  int GridSize = atoi(argv[2]);
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);
  
  matSum<<< gridDim, blockDim >>>(Pd, Md, Nd, Width);

  // copy result from device to host
  hipMemcpy(P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost);

  // clean up memory  
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);

  end = clock();
  double time_taken = (double)(end - start) / CLOCKS_PER_SEC;
  printf("%.8f", time_taken);

  free(M);
  free(N);
  free(P);

  return 0;
}