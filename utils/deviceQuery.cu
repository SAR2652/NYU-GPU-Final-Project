#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main()
{
  hipError_t error;
  hipDeviceProp_t dev;
  int dev_cnt = 0;

hipProfilerStart();
  // return device numbers with compute capability >= 1.0
  error = hipGetDeviceCount (&dev_cnt);
  if(error != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  // Get properties of each device
  for(int i = 0; i < dev_cnt; i++)
  {
     error = hipGetDeviceProperties(&dev, i);
     if(error != hipSuccess)
     {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
     }
     printf("Device ID=%d\n", i);
     printf("Device Name=%s\n",dev.name);
     printf("Compute Capability=%d.%d\n",dev.major, dev.minor);
     printf("Total Global Memory=%ld\n", dev.totalGlobalMem/1024);
     printf("Shared Memory per Block=%d\n",dev.sharedMemPerBlock);
     printf("Registers per Block=%d\n", dev.regsPerBlock);
     printf("Warp Size=%d\n", dev.warpSize);
     printf("Maximum Threads per Block=%d\n",dev.maxThreadsPerBlock);
     printf("Thread Dimension Z=%d\n", dev.maxThreadsDim[0]);
     printf("Thread Dimension Y=%d\n", dev.maxThreadsDim[1]);
     printf("Thread Dimension X=%d\n", dev.maxThreadsDim[2]);
     printf("Grid Size Z=%d\n", dev.maxGridSize[2]);
     printf("Grid Size Y=%d\n", dev.maxGridSize[1]);
     printf("Grid Size X=%d\n", dev.maxGridSize[0]);
     printf("Clock Rate=%d\n",dev.clockRate);
     printf("Total Constant Memory=%ld\n",dev.totalConstMem);
     printf("Multiprocessor Count=%d\n",dev.multiProcessorCount);
     printf("integrated=%d\n",dev.integrated);
     printf("Asynchronous Engine Count=%d\n",dev.asyncEngineCount);
     printf("Memory Bus Width=%d\n",dev.memoryBusWidth);
     printf("Memory Clock Rate=%d\n",dev.memoryClockRate);
     printf("L2 Cache Size=%d\n", dev.l2CacheSize);
     printf("Maximum Threads per Multiprocessor=%d\n", dev.maxThreadsPerMultiProcessor);
     printf("Concurrent Kernels=%d\n", dev.concurrentKernels);
  }

hipProfilerStop();
  return 0;

}
